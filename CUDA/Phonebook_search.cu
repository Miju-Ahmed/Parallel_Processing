/*
https://colab.research.google.com/drive/1MXExWMxIUQVB0JyCUD40FJzE796ksfP9#scrollTo=Ecz-fVBtY5-v
command -> 
!nvcc -arch=sm_75 phonebook_search.cu -o miju
!time ./miju AKHTER 2 > out.txt
*/

// %%writefile phonebook_search.cu
#include <bits/stdc++.h>
using namespace std;
#include <hip/hip_runtime.h>

struct Contact{
    char name[65];
    char phone_number[65];
};


string getInput(ifstream& file){
    string ans;
    char c;
    int insideQuote = 0;
    while(file.get(c)){
        if(c == '\"'){
            if(insideQuote == 1) break;
            insideQuote = 1;
        }else{
            if(insideQuote){
                ans.push_back(c);
            }
        }
    }
    return ans;
}

__device__ bool check(char* str1, char* str2){
    for(int i = 0; str1[i] != '\0'; i++){
        int flag = 1;
        for(int j = 0; str2[j] != '\0' ; j++){
            if(str1[i + j] != str2[j]){
                flag = 0;
                break;
            }
        }
        if(flag == 1) return true;
    }
    return false;
}


// __global__ void myKernel(Contact* phoneBook, char* pat, int offset){
//     int threadNumber = threadIdx.x + offset;
//     if(check(phoneBook[threadNumber].name, pat)){
//         printf("%s %s\n", phoneBook[threadNumber].name, phoneBook[threadNumber].phone_number);
//     }
// }

__global__ void FindPhone(Contact* phoneBook, char* pat, int totalSize) {
    int threadNumber = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadNumber < totalSize) {
        if (check(phoneBook[threadNumber].name, pat)) {
            printf("%s %s\n", phoneBook[threadNumber].name, phoneBook[threadNumber].phone_number);
        }
    }
}



int main(int argc, char* argv[])
{
    int threadLimit = atoi(argv[2]);

    ifstream myfile("/home/cse/Desktop/Noise_Resilant_Build_Model/MPI/labtest_dataset1.txt");
    vector<Contact> phoneBook;

    // int count = 0;

    while(myfile.peek() != EOF){

        // if(count > 100000) break;
        // count++;

        string name = getInput(myfile);
        string phoneNum = getInput(myfile);

        Contact c;
        strcpy(c.name, name.c_str());
        strcpy(c.phone_number, phoneNum.c_str());

        phoneBook.push_back(c);
    }

    string search_name = argv[1];
    char pat[65];
    strcpy(pat, search_name.c_str());


    char* d_pat;
    hipMalloc(&d_pat, 65); //memory allocation
    hipMemcpy(d_pat, pat, 65, hipMemcpyHostToDevice); //copying to device

    int n = phoneBook.size();
    Contact* d_phoneBook;
    hipMalloc(&d_phoneBook, n*sizeof(Contact));
    hipMemcpy(d_phoneBook, phoneBook.data(), n * sizeof(Contact), hipMemcpyHostToDevice);


    // int bakiAche = n;
    // int offset = 0;
    // while(bakiAche > 0){

    //     int batchSize = min(threadLimit, bakiAche);
    //     myKernel<<<1,batchSize>>>(d_phoneBook, d_pat, offset);
    //     cudaDeviceSynchronize();

    //     bakiAche -= batchSize;
    //     offset += batchSize;
    // }

    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    FindPhone<<<blocks, threadsPerBlock>>>(d_phoneBook, d_pat, n);
    hipDeviceSynchronize();

}
