#include "hip/hip_runtime.h"
/*
Command for run
!nvcc -arch=sm_75 Miju.cu -o miju
!time ./miju AKASH 50 > output.txt
*/

%%writefile Miju.cu
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

struct Contact {
    char name[65];
    char phone_number[65];
};

// Utility: Trim whitespace
string trim(const string& str) {
    size_t start = str.find_first_not_of(" \t");
    if (start == string::npos) return "";
    size_t end = str.find_last_not_of(" \t");
    return str.substr(start, end - start + 1);
}

// Extract name between two //
string getInput(ifstream& file) {
    string line;
    getline(file, line);

    size_t first_delim = line.find("//");
    if (first_delim == string::npos) return "";

    size_t second_delim = line.find("//", first_delim + 2);
    if (second_delim == string::npos) return "";

    return trim(line.substr(first_delim + 2, second_delim - (first_delim + 2)));
}

// Extract phone number after last //
string getPhoneNumber(ifstream& file) {
    string line;
    getline(file, line);

    size_t delim = line.rfind("//");
    if (delim == string::npos) return "";

    return trim(line.substr(delim + 2));
}

__device__ bool check(char* str1, char* str2) {
    for (int i = 0; str1[i] != '\0'; i++) {
        int flag = 1;
        for (int j = 0; str2[j] != '\0'; j++) {
            if (str1[i + j] == '\0' || str1[i + j] != str2[j]) {
                flag = 0;
                break;
            }
        }
        if (flag == 1) return true;
    }
    return false;
}

__global__ void myKernel(Contact* phoneBook, char* pat, int offset, Contact* matches, int* matchCount) {
    int threadNumber = threadIdx.x + offset;
    if (check(phoneBook[threadNumber].name, pat)) {
        int idx = atomicAdd(matchCount, 1);
        matches[idx] = phoneBook[threadNumber];
    }
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        cerr << "Usage: " << argv[0] << " <search_name> <thread_limit>" << endl;
        return 1;
    }

    int threadLimit = atoi(argv[2]);
    string search_name = argv[1];

    ifstream myfile("/content/drive/MyDrive/labtest_dataset.txt");
    if (!myfile.is_open()) {
        cerr << "Error opening file!" << endl;
        return 1;
    }

    vector<Contact> phoneBook;
    string line;
    int count = 0;

    while (getline(myfile, line)) {
        if (line.empty()) continue;
        if (++count > 200000) break;

        string name = getInput(myfile);
        string phoneNum = getPhoneNumber(myfile);

        Contact c;
        strncpy(c.name, name.c_str(), sizeof(c.name) - 1);
        c.name[sizeof(c.name) - 1] = '\0';
        strncpy(c.phone_number, phoneNum.c_str(), sizeof(c.phone_number) - 1);
        c.phone_number[sizeof(c.phone_number) - 1] = '\0';

        phoneBook.push_back(c);
    }

    char pat[65];
    strncpy(pat, search_name.c_str(), sizeof(pat) - 1);
    pat[sizeof(pat) - 1] = '\0';

    // Allocate memory on device
    int n = phoneBook.size();
    Contact* d_phoneBook;
    char* d_pat;
    Contact* d_matches;
    int* d_matchCount;

    hipMalloc(&d_phoneBook, n * sizeof(Contact));
    hipMemcpy(d_phoneBook, phoneBook.data(), n * sizeof(Contact), hipMemcpyHostToDevice);

    hipMalloc(&d_pat, 65);
    hipMemcpy(d_pat, pat, 65, hipMemcpyHostToDevice);

    hipMalloc(&d_matches, n * sizeof(Contact)); // Worst case: all match
    hipMalloc(&d_matchCount, sizeof(int));
    hipMemset(d_matchCount, 0, sizeof(int));

    int remaining = n;
    int offset = 0;
    while (remaining > 0) {
        int batchSize = min(threadLimit, remaining);
        myKernel<<<1, batchSize>>>(d_phoneBook, d_pat, offset, d_matches, d_matchCount);
        hipDeviceSynchronize();
        remaining -= batchSize;
        offset += batchSize;
    }

    // Get match count
    int h_matchCount;
    hipMemcpy(&h_matchCount, d_matchCount, sizeof(int), hipMemcpyDeviceToHost);

    // Copy matches back
    vector<Contact> matches(h_matchCount);
    hipMemcpy(matches.data(), d_matches, h_matchCount * sizeof(Contact), hipMemcpyDeviceToHost);

    // Sort matches by name
    sort(matches.begin(), matches.end(), [](const Contact& a, const Contact& b) {
        return strcmp(a.name, b.name) < 0;
    });

    // Print sorted matches
    for (const auto& c : matches) {
        cout << c.name << " " << c.phone_number << endl;
    }

    // Cleanup
    hipFree(d_phoneBook);
    hipFree(d_pat);
    hipFree(d_matches);
    hipFree(d_matchCount);

    return 0;
}
